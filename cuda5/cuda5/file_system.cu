#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

 //__device__ __managed__ u32 gtime = 0;

__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS,
							uchar* dir, int DIR_SIZE, int DIR_ENTRIES)
{
  // init variables
  fs->volume = volume;
  fs->dir = dir;
  // init constants
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;
  fs->FCB_SIZE = FCB_SIZE;
  fs->FCB_ENTRIES = FCB_ENTRIES;
  fs->STORAGE_SIZE = VOLUME_SIZE;
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;
  fs->MAX_FILE_NUM = MAX_FILE_NUM;
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;
	  
  for (int i = 0; i < fs->SUPERBLOCK_SIZE / 8; i++) {
	  fs->volume[i] = 0b11111111;
  }
  for (int i = 0; i < 1024; i++) {
	  fs->dir[1024 - i] = 0b11111111;
  }
  fs->DIR_SIZE = FCB_SIZE;
  fs->DIR_ENTRIES = FCB_ENTRIES;
  mkdir(fs, "root");
}

__device__ int pow(int base, int power) {			// Pow function
	if (power == 0) return 1;
	int result = base;
	for (int i = 0; i < power-1; i++) {
		result *= base;
	}
	return result;
}

__device__ int strlen(char* str) {					// Return the length of string
	int i = 0;
	while (*(str + i) != '\0') {
		i += 1;
		if (i > 21)
			return 21;
	}
	return i;
}

__device__ void compact(FileSystem* fs, u32 file_pointer) {

	printf("[OP] Compact \n");

	uchar* file = &fs->volume[fs->SUPERBLOCK_SIZE + file_pointer * fs->FCB_SIZE];
	int fragment = *(int*)(file + 28) * 32;
	int fragment_size = *(int*)(file + 24);
	if (fragment_size % 32 != 0) fragment_size = (fragment_size / fs->STORAGE_BLOCK_SIZE + 1) * fs->STORAGE_BLOCK_SIZE;
	int bit = (fragment + fragment_size)/32;									// Position in bit map

	if (bit < fs->SUPERBLOCK_SIZE) {
		if ((fs->volume[bit / 8] >> (7 - bit % 8)) % 2 == 0) {					// Modify storage block address in FCB
			for (int i = 0; i < fs->FCB_ENTRIES; i++) {
				if (*(int*)&fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 28] > *(int*)(file+28)) {
					*(int*)&fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + 28] -= fragment_size/32;
				}
			}
		}
		while (true) {												
			if ((fs->volume[bit / 8] >> (7 - bit % 8)) % 2 == 0) {				// Check if the block is free, if so, it's end
				for (int i = 0; i < 32; i++) {									// else, move contents forwards
					fs->volume[fragment + fs->FILE_BASE_ADDRESS + i] = fs->volume[fragment + fs->FILE_BASE_ADDRESS + i + fragment_size];
				}
				fragment += 32;
				bit += 1;
			}
			else break;
		}
	}
	/* free ending bit map */
	for (int j = 0; j < fragment_size / fs->STORAGE_BLOCK_SIZE; j++) {
		fs->volume[(fragment / fs->STORAGE_BLOCK_SIZE + j) / 8] += pow(2,(7 - (fragment / fs->STORAGE_BLOCK_SIZE + j) % 8));
	}
	for (int i = 0; i < fragment_size; i++) {				// Reset the storage
		fs->volume[fs->FILE_BASE_ADDRESS + fragment + i] = '\0';
	}
	*(int*)(file + 28) = fragment / 32;											// Re-allocate block
	printf("[OP] Compact Finish! \n");
}

__device__ u32 fs_open(FileSystem *fs, char *filename, int operation) {

	printf("\n[OP] File open: %s \n",filename);

	/* Check the length of filename */
	if (strlen(filename) > 20) {
		printf("[ERROR] File name too long");
	}

	u32 block_num = 0;
	int file_pointer = 0;
	bool file_exist = false;

	/* Search the file in FCB */
	for (int i = 0; i < fs->FCB_ENTRIES; i++) {
		int j = 0;
		while (fs->volume[fs->SUPERBLOCK_SIZE + 32 * i + j] != '\0') {
			file_exist = true;
			if (fs->volume[fs->SUPERBLOCK_SIZE + 32 * i + j] != *(filename + j)) {
				file_exist = false;
				break;
			}
			j += 1;
		}
		if (file_exist == true) {
			file_pointer = i;
			break;
		}
	}
	/* If the file does not exist, create a new one */
	if (!file_exist) {																		
		if (operation == G_READ) {																			// Cannot read the file
			printf("[ERROR] No such file! Cannot read it!");
			return -1;
		}
		else if (operation == G_WRITE) {
			if (fs->current_file_num >= fs->MAX_FILE_NUM) {													
				printf("[ERROR] FCB is full!\n");															// Check if the FCB is full
				return -1;
			}
			//printf("[INFO] Current file num %d\n",fs->current_file_num);
			int i, j;
			bool free_flag = false;
			for (i = 0; i < fs->SUPERBLOCK_SIZE / 8; i++) {													// Find free block
				for (j=7; j >= 0; j--) {
					int k = (fs->volume[i] >> j);
					if (k % 2 == 1) {
						free_flag = true;
						break;
					}
				}
				if (free_flag) break;
			}
			fs->volume[i] -= pow(2, j);																		// Mark the bit as occupied
			block_num = (8 * i + 7 - j);
			printf("[INFO] File not exist\n");
			printf("[INFO] ------------------------ New block number %d\n", block_num);

			for (int i = 0; i < fs->FCB_ENTRIES; i++) {										
				if (fs->volume[fs->SUPERBLOCK_SIZE + 32 * i] == '\0') {
					file_pointer = i;
					break; 
				}
			}

			for (int count = 0; count < strlen(filename); count++) {										// Copy file name
				fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer + count] = (uchar)*(filename + count);
			}
			*(u32*)&fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer + 20] = fs->time;			// Record mod time
			*(int*)&fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer + 24] = 0;				// Default size 32
			*(int*)&fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer + 28] = block_num;			// Block address
			fs->current_file_num += 1;
		}
	}

	printf("[OP] File successfully open! \n");
	return file_pointer;
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 read_size, u32 file_pointer) {

	printf("\n[OP] Read file: %s\n",&fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer]);

	fs->time += 1;

	if (/*!read permission*/0) {																			// User root always has permission
		return;
	}
	if (read_size > *(int*)(&fs->volume[fs->SUPERBLOCK_SIZE + file_pointer * fs->FCB_SIZE]+24)) {
		printf("[ERROR] Read size greater than file size!\n");
		return;
	}
	for (int i = 0; i < read_size; i++) {
		int file_addr = fs->FILE_BASE_ADDRESS + *(int*)&fs->volume[fs->FCB_SIZE * file_pointer + fs->SUPERBLOCK_SIZE + 28] * fs->STORAGE_BLOCK_SIZE;
		*(output + i) = fs->volume[file_addr + i];
	}
}

__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 input_size, u32 file_pointer) {

	printf("\n[OP] Write %s  Size %d \n", &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer],input_size);
	if (input_size > 1024) printf("[ERROR] Exceed max file size!\n");
	fs->time += 1;

	uchar* file = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer];
	*(int*)(file +20) = fs->time;
	if (/*!write permission*/0) {
		return;
	}
	int original_block_num = 1;
	if (*(int*)(file + 24) > 0) original_block_num = (*(int*)(file + 24) - 1) / 32;
	if ((input_size-1)/32 != original_block_num) {			// If the input size occupies different #blocks with original file compact
		compact(fs, file_pointer);									// reallocate and write
		*(int*)(file + 24) = input_size;
		return fs_write(fs, input, input_size, file_pointer);		// recursion print write twice...
	}
	*(int*)(file + 24) = input_size;
	int file_addr = fs->FILE_BASE_ADDRESS + *(int*)&fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * file_pointer + 28] * fs->STORAGE_BLOCK_SIZE;

	for (int i = 0; i < input_size; i++) {		
		fs->volume[file_addr + i] =  *(input + i);

		if (i % fs->STORAGE_BLOCK_SIZE == 0) {						// Occupy bit map
			int bit = *(int*)(file + 28) + i/32;
			if ((fs->volume[bit / 8] >> (7 - bit % 8)) % 2 == 1) {
				fs->volume[bit / 8] -= pow(2, (7 - bit % 8));

			}
		}
	}

	printf("[OP] Write successfully! \n");
	return 0;
}

__device__ void fs_gsys(FileSystem *fs, int operation) {

	printf("\n[OP] List ");
	if (operation == LS_D) printf("LS_D ");
	else printf("LS_S ");
	printf("\n");

	/* gather all files */
	int* files = new int[1024];							// stack overflow on my computer， move to heap x_x
	int sort_item = 20;
	int count = 0;
	for (int i = 0; i < fs->FCB_ENTRIES; i++) {
		int fcb_pos = 32 * i;
		if (fs->volume[fs->SUPERBLOCK_SIZE + fcb_pos] != '\0') {
			files[count] = fcb_pos;
			count += 1;
		}
	}

	/* Sort */
	printf("Sort..");
	if (operation == LS_S) sort_item = 24;			// Sort_item(addr offset in fcb), indicates the type of sort
	for (int i = 0; i < count; i++) {
		for (int j = i + 1; j < count; j++) {
			if (*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + sort_item)\
				< *(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[j]]) + sort_item)) {
				int temp = files[j];
				files[j] = files[i];
				files[i] = temp;
			}
			else if (sort_item == 24 && *(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]])\
				+ sort_item) == *(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[j]]) + sort_item)) {

				if (*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + 20) < *(int*)\
					(&(fs->volume[fs->SUPERBLOCK_SIZE + files[j]]) + 20)) {
					int temp = files[j];
					files[j] = files[i];
					files[i] = temp;
				}
			}
		}
	}

	printf("Display..\n");
	printf(" Filename | Mod time | Size | Owner | Permission |\n");
	for (int i = 0; i < count; i++) {
		printf("%s          %d       %d      root     rwe\n",
			(char*)&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]),
			*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + 20),
			*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + 24));
	}
}

__device__ void fs_gsys(FileSystem *fs, int operation, char *filename) {

	printf("\n[OP] Remove %s \n",filename);

	u32 file_pointer;
	uchar* file;
	bool file_exist = false;
	int i;
	int j = 0;
	for (i = 0; i < fs->FCB_ENTRIES; i++) {
		if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i] == '\0') continue;			// Find the file
		printf("[INFO] Filename match! ");
		while (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + j] != '\0') {
			file_exist = true;
			printf("%c", fs->volume[fs->SUPERBLOCK_SIZE + 32 * i + j]);
			if (fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_SIZE * i + j] != *(filename + j)) {
				file_exist = false;
				break;
			}
			j += 1;
		}
		printf("\n");
		if (file_exist == true) {
			file_pointer = i;
			file = &fs->volume[fs->SUPERBLOCK_SIZE + 32 * file_pointer];
			break;
		}
	}

	if (!file_exist) {
		printf("[ERROR] No such file! Cannot remove!");
		return;
	}
	else if (operation < RM) {
		// List this file and print you give a list operation
		printf(" Filename | Mod time | Size | Owner | Permission |\n");
		printf("%s          %d       %d      root     rwe\n",
			file, *(int*)(file + 24), *(int*)(file + 20));
	}
	else {
		compact(fs, *(int*)(file + 28));
		int file_block_size = *(int*)(file + 24);
		if (file_block_size % 32 != 0) file_block_size = (file_block_size / fs->STORAGE_BLOCK_SIZE + 1) * fs->STORAGE_BLOCK_SIZE;

		*file = '\0';											// Free the fcb block
		fs->current_file_num -= 1;
	}
	printf("\n[OP] Successfully Removed \n");
}

__device__ void mkdir(FileSystem* fs, char* dir_name) {
	if (fs->current_dir == -1) {
		*(short *)&fs->dir[0] = -1;																		// dir
		for (int count = 0; count < strlen(dir_name); count++) {										// Copy file name
			fs->dir[2 + count] = *(dir_name + count);
		}	
		u32 fp = fs_open(fs, dir_name, G_WRITE);
		*(short *)&fs->dir[22] = 0;																		// file_num
		*(short *)&fs->dir[24] = -1;																		// next
		*(short *)&fs->dir[26] = fp;																		// file_pointer
		*(short *)&fs->dir[28] = -1;																		// child_pointer
		*(short *)&fs->dir[30] = -1;																		// parent_pointer
	}
	else {
		short dir_file = *(short*)&fs->dir[fs->current_dir + 28];
		if (dir_file == -1) {
			dir_file = fs->current_dir;
		}
		else {
			while (true) {
				if (*(short*)&fs->dir[dir_file] != -1) {			// have next file/dir
					if (*(short*)&fs->dir[dir_file + 2] != -1)
						dir_file = *(short*)&fs->dir[dir_file + 2];							// to next node
					else break;
				}
				else {
					int j = 0;
					bool match = false;
					while (fs->dir[dir_file + 2 + j] != '\0') {
						match = true;
						if (fs->dir[dir_file + 2 + j] != *(dir_name + j)) {
							match = false;
							break;
						}
						j += 1;
					}
					if (match) {
						printf("Dir %s already exist!", dir_name);
						return;
					}
					else {
						if (*(short*)&fs->dir[dir_file + 24] != -1)
							dir_file = *(short*)&fs->dir[dir_file + 24];
						else break;
					}
				}
			}
		}

		// now : [ fcb_addr | -1 ]  -> [ | ]

		int i, j;
		int free_num = 0;
		bool free_flag = false;
		for (i = 0; i < 1024; i++) {													// Find free block
			for (j = 7; j >= 0; j--) {
				int k = (fs->dir[fs->DIR_ENTRIES * fs->DIR_SIZE + i] >> j);
				if (k % 2 == 1) free_num += 1;
				else free_num = 0;
				if (free_num == 4) {
					free_flag = true;
					break;
				}
			}
			if (free_flag) break;
		}
		for (int a = 0; a < 4; a++) {
			fs->dir[fs->DIR_ENTRIES * fs->DIR_SIZE + i] -= pow(2, (j + a ) % 8);					// Mark the bit as occupied
		}
		int dir_block_num = (8 * i + 7 - j) - 3;
		if (*(short*)&fs->dir[dir_file] != -1)
			*(short*)&fs->dir[dir_file + 2] = dir_block_num;
		else
			*(short*)&fs->dir[dir_file + 24] = dir_block_num;

		*(short*)&fs->dir[dir_block_num] = -1;
		for (int count = 0; count < strlen(dir_name); count++) {										// Copy file name
			fs->dir[dir_block_num + 2 + count] = *(dir_name + count);
		}
		*(short *)&fs->dir[dir_block_num + 22] = 0;
		//if (*(short *)&fs->dir[fs->current_dir + 24] + 1 <= 3)
		*(short *)&fs->dir[dir_block_num + 24] = -1;														// next
		u32 fp = fs_open(fs, dir_name, G_WRITE);
		*(short *)&fs->dir[dir_block_num + 26] = fp;														// file_pointer
		*(short *)&fs->dir[dir_block_num + 28] = -1;																		// child_pointer
		*(short *)&fs->dir[dir_block_num + 30] = fs->current_dir;															// parent_pointer
	}
}

__device__ void cd(FileSystem* fs, int operation, char* dir_name) {
	if (operation != CD) return;
	int dir = 0;
	if (*(short*)&fs->dir[fs->current_dir + 28] != -1) {
		dir = *(short*)&fs->dir[fs->current_dir + 28];
	}
	else {
		printf("[ERROR] No sub-directory!");
	}
	while (true) {
		if (*(short*)&fs->dir[dir] == -1) {
			int j = 0;
			bool match = false;
			while (fs->dir[dir + 2 + j] != '\0') {
				match = true;
				if (fs->dir[dir + 2 + j] != *(dir_name + j)) {
					match = false;
					break;
				}
				j += 1;
			}
			if (match) {
				fs->current_dir = dir;
				break;
			}
			else {
				dir = fs->dir[dir + 24];
				if (dir == -1) {
					printf("[ERROR] No such directory!");
				}
			}
		}
		else {
			dir = *(short*)&fs->dir[dir + 2];
			if (dir == -1) {
				printf("[ERROR] No such directory!");
			}
		}
	}
}
__device__ void cd_p(FileSystem* fs, int operation) {
	if (operation != CD_P) return;
	fs->current_dir = *(short*)&fs->dir[fs->current_dir + 30];
}

__device__ void cd_root(FileSystem* fs, int operation, char* dir_name) {
	if (operation != CD_ROOT) return;
	fs->current_dir = 0;
}
__device__ void fs_gsys1(FileSystem *fs, int operation) {

	printf("\n[OP] List ");
	if (operation == LS_D) printf("LS_D ");
	else printf("LS_S ");
	printf("\n");

	/* gather all files */
	int* files = new int[50];							// stack overflow on my computer， move to heap x_x
	int sort_item = 20;
	int count = 0;
	int dir = -1;
	if (*(short*)&fs->dir[fs->current_dir + 28] == -1) return;
	dir = *(short*)&fs->dir[fs->current_dir + 28];

	while (true) {
		if (*(short*)&fs->dir[dir] == -1) {
			files[count] = *(short*)&fs->dir[dir];
			if (*(short*)&fs->dir[dir + 28] == -1) break;
			dir = *(short*)&fs->dir[dir + 28];
		}
		else {
			files[count] = *(short*)&fs->dir[dir];
			if (*(short*)&fs->dir[dir + 2] == -1) break;
			dir = *(short*)&fs->dir[dir + 2];
			count += 1;
		}
	}

	/* Sort */
	printf("Sort..");
	if (operation == LS_S) sort_item = 24;			// Sort_item(addr offset in fcb), indicates the type of sort
	for (int i = 0; i < count; i++) {
		for (int j = i + 1; j < count; j++) {
			if (*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + sort_item)\
				< *(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[j]]) + sort_item)) {
				int temp = files[j];
				files[j] = files[i];
				files[i] = temp;
			}
			else if (sort_item == 24 && *(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]])\
				+ sort_item) == *(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[j]]) + sort_item)) {

				if (*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + 20) < *(int*)\
					(&(fs->volume[fs->SUPERBLOCK_SIZE + files[j]]) + 20)) {
					int temp = files[j];
					files[j] = files[i];
					files[i] = temp;
				}
			}
		}
	}

	printf("Display..\n");
	printf(" Filename | Mod time | Size | Owner | Permission |\n");
	for (int i = 0; i < count; i++) {
		printf("%s          %d       %d      root     rwe",
			(char*)&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]),
			*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + 20),
			*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + 24));
		if (*(int*)(&(fs->volume[fs->SUPERBLOCK_SIZE + files[i]]) + 20) == 0) printf(" D \n");
		else printf(" D \n");
	}
}


__device__ void pwd(FileSystem* fs, int operation) {
	if (operation != PWD) return;
	int dirs[3];
	int dir = fs->current_dir;
	int count = 0;
	while (true) {
		dirs[count] = dir;
		if (*(short*)&fs->dir[dir + 30] == -1) return;
		dir = *(short*)&fs->dir[dir + 30];
		count += 1;
	}

	for (count; count >= 0; count--) {
		printf("/");
		printf("%s", *(char*)fs->dir[dirs[count]]);
	}
	printf("/\n");
}

__device__ void rmrf(FileSystem* fs, int operation, char* dir_name) {
	if (operation != RM_RF) return;
	int dir = 0;
	if (*(short*)&fs->dir[fs->current_dir + 28] != -1) {
		dir = *(short*)&fs->dir[fs->current_dir + 28];
	}
	else {
		printf("[ERROR] No sub-directory!");
	}
	while (true) {
		if (*(short*)&fs->dir[dir] == -1) {
			int j = 0;
			bool match = false;
			while (fs->dir[dir + 2 + j] != '\0') {
				match = true;
				if (fs->dir[dir + 2 + j] != *(dir_name + j)) {
					match = false;
					break;
				}
				j += 1;
			}
			if (match) {
				int file = fs->dir[dir + 28];
				if (fs->dir[file + 28] != -1) {
					while (true) {
						if (*(short*)&fs->dir[file] == -1) {
							fs->current_dir = file;
							rmrf(fs, RM_RF, (char*)(&fs->dir[file + 2]));
							fs->current_dir = dir;
							if (*(short*)&fs->dir[file + 24] != -1) {
								file = *(short*)&fs->dir[file + 24];
							}
							else break;
						}
						else {
							fs_gsys(fs, RM, (char*)fs->volume[*(short*)&fs->dir[file]]);
							if (fs->dir[file + 2] != -1)file = *(short*)&fs->dir[file + 2];
							else break;
						}
					}
				}
				fs_gsys(fs, RM, (char*)fs->dir[dir + 2]);
				if (fs->dir[fs->dir[dir + 30] + 28] != -1)
					fs->dir[fs->dir[dir + 30] + 28] = fs->dir[fs->dir[dir + 30] + 24];


			}
			else {
				dir = fs->dir[dir + 24];
				if (dir == -1) {
					printf("[ERROR] No such directory!");
				}
			}
		}
		else {
			dir = *(short*)&fs->dir[dir + 2];
			if (dir == -1) {
				printf("[ERROR] No such directory!");
			}
		}
	}
}




